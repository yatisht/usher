#include "hip/hip_runtime.h"
// This program performs a simple single-precision Ax+Y operation
// using cudaGraph and verifies its result.

#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cudaflow.hpp>

// Kernel: saxpy
__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a*x[i] + y[i];
  }
}

// Function: main
int main() {
  
  const unsigned N = 1<<20;

  std::vector<float> hx, hy;

  float* dx {nullptr};
  float* dy {nullptr};
  
  // allocate x
  hx.resize(N, 1.0f);
  hipMalloc(&dx, N*sizeof(float));

  // allocate y
  hy.resize(N, 2.0f);
  hipMalloc(&dy, N*sizeof(float));
  
  // saxpy cudaGraph
  tf::cudaGraph cg;
  auto h2d_x = cg.copy(dx, hx.data(), N);
  auto h2d_y = cg.copy(dy, hy.data(), N);
  auto d2h_x = cg.copy(hx.data(), dx, N);
  auto d2h_y = cg.copy(hy.data(), dy, N);
  auto kernel = cg.kernel((N+255)/256, 256, 0, saxpy, N, 2.0f, dx, dy);
  kernel.succeed(h2d_x, h2d_y)
        .precede(d2h_x, d2h_y);
  
  tf::cudaStream stream;
  tf::cudaGraphExec exec(cg);
  
  stream.run(exec)
        .synchronize();
  
  // visualize this cudaflow
  cg.dump(std::cout);

  // Add a verification task
  float max_error = 0.0f;
  for (size_t i = 0; i < N; i++) {
    max_error = std::max(max_error, abs(hx[i]-1.0f));
    max_error = std::max(max_error, abs(hy[i]-4.0f));
  }
  std::cout << "saxpy finished with max error: " << max_error << '\n';

  // free memory
  hipFree(dx);
  hipFree(dy);

  tf::cudaGraph cg2(std::move(cg));
  tf::cudaGraphExec exec2(std::move(exec));

  return 0;
}


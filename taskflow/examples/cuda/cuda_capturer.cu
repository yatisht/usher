#include "hip/hip_runtime.h"
// This program performs a simple single-precision Ax+Y operation
// using a cudaFlow capturer and verifies its result.

#include <taskflow/cuda/cudaflow.hpp>

// Kernel: saxpy
__global__ void saxpy(int n, float a, float *x, float *y) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) {
    y[i] = a*x[i] + y[i];
  }
}

// Function: main
int main() {
  
  const unsigned N = 1<<20;

  std::vector<float> hx(N, 1.0f), hy(N, 2.0f);

  auto dx = tf::cuda_malloc_device<float>(N);
  auto dy = tf::cuda_malloc_device<float>(N);

  tf::cudaFlowCapturer cf;

  auto h2d_x  = cf.copy(dx, hx.data(), N).name("h2d_x");
  auto h2d_y  = cf.copy(dy, hy.data(), N).name("h2d_y");
  auto d2h_x  = cf.copy(hx.data(), dx, N).name("d2h_x");
  auto d2h_y  = cf.copy(hy.data(), dy, N).name("d2h_y");
  auto kernel = cf.kernel((N+255)/256, 256, 0, saxpy, N, 2.0f, dx, dy)
                  .name("saxpy");
  kernel.succeed(h2d_x, h2d_y)
        .precede(d2h_x, d2h_y);

  // execute the cudaflow capturer
  std::cout << "running cudaflow capturer ...\n";
  tf::cudaStream stream;
  auto exec = cf.instantiate();
  exec.run(stream);
  stream.synchronize();

  // inspect the result
  float max_error = 0.0f;
  for (size_t i = 0; i < N; i++) {
    max_error = std::max(max_error, abs(hx[i]-1.0f));
    max_error = std::max(max_error, abs(hy[i]-4.0f));
  }
  std::cout << "saxpy finished with max error: " << max_error << '\n';

  // free memory
  tf::cuda_free(dx);
  tf::cuda_free(dy);
  
  // dump the cudaFlow graph
  cf.dump(std::cout);

  // dump the native CUDA graph
  cf.dump_native_graph(std::cout);

  return 0;
}


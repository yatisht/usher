#include "hip/hip_runtime.h"
#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cudaflow.hpp>

int main() {

  // CUDA version
  std::cout << "========================================\n"
            << "CUDA version: " 
            << tf::cuda_get_runtime_version() << '\n'
            << "CUDA driver version: " 
            << tf::cuda_get_driver_version() << '\n';

  // Number of CUDA devices
  auto num_cuda_devices = tf::cuda_get_num_devices();

  std::cout << "There are " << num_cuda_devices << " CUDA devices.\n";

  // Iterate each device and dump its property
  std::cout << "\nquerying device properties ...\n";
  for(size_t i = 0; i < num_cuda_devices; ++i) {
    std::cout << "CUDA device #" << i << '\n';
    tf::cuda_dump_device_property(std::cout, tf::cuda_get_device_property(i));
  }

  // we can also query each device property attribute by attribute
  std::cout << "\nquerying device attributes ...\n";
  for(size_t i = 0; i< num_cuda_devices; ++i) {
    std::cout << "CUDA device #" << i << '\n';
    std::cout << "Compute capability   : "
              << tf::cuda_get_device_compute_capability_major(i) << '.'
              << tf::cuda_get_device_compute_capability_minor(i) << '\n';
    std::cout << "max threads per block: "
              << tf::cuda_get_device_max_threads_per_block(i) << '\n'
              << "max x-dim   per block: "
              << tf::cuda_get_device_max_x_dim_per_block(i) << '\n'
              << "max y-dim   per block: "
              << tf::cuda_get_device_max_y_dim_per_block(i) << '\n'
              << "max z-dim   per block: "
              << tf::cuda_get_device_max_z_dim_per_block(i) << '\n'
              << "max x-dim   per grid : "
              << tf::cuda_get_device_max_x_dim_per_grid(i) << '\n'
              << "max y-dim   per grid : "
              << tf::cuda_get_device_max_y_dim_per_grid(i) << '\n'
              << "max z-dim   per grid : "
              << tf::cuda_get_device_max_z_dim_per_grid(i) << '\n'
              << "warp size            : "
              << tf::cuda_get_device_warp_size(i) << '\n'
              << "unified addressing?  : "
              << tf::cuda_get_device_unified_addressing(i) << '\n';
  }

  return 0;
}




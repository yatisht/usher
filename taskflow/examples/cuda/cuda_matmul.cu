#include "hip/hip_runtime.h"
// The example shows how to use cudaFlow to multiply two 2D matrices.

#include <taskflow/taskflow.hpp>
#include <taskflow/algorithm/for_each.hpp>
#include <taskflow/cuda/cudaflow.hpp>

// Kernel: matmul
__global__ void matmul(int *a, int *b, int *c, int m, int n, int k) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int sum = 0;
  if(col < k && row < m) {
    for(int i = 0; i < n; i++) {
      sum += a[row * n + i] * b[i * k + col];
    }
    c[row * k + col] = sum;
  }
}

// Matrix multiplication using GPU
auto gpu(int M, int N, int K) {
  
  std::vector<int> ha, hb, hc;
  int *da, *db, *dc;

  tf::Taskflow taskflow("MatrixMultiplication");
  tf::Executor executor;

  // allocate the host and device storage for a
  auto allocate_a = taskflow.emplace([&](){
    ha.resize(M*N, M+N);
    TF_CHECK_CUDA(hipMalloc(&da, M*N*sizeof(int)), "failed to allocate a");
  }).name("allocate_a");
  
  // allocate the host and device storage for b
  auto allocate_b = taskflow.emplace([&](){
    hb.resize(N*K, N+K);
    TF_CHECK_CUDA(hipMalloc(&db, N*K*sizeof(int)), "failed to allocate b");
  }).name("allocate_b");
  
  // allocate the host and device storage for c
  auto allocate_c = taskflow.emplace([&](){
    hc.resize(M*K);
    TF_CHECK_CUDA(hipMalloc(&dc, M*K*sizeof(int)), "failed to allocate c");
  }).name("allocate_c");
  
  // create a cudaFlow to run the matrix multiplication
  auto cudaFlow = taskflow.emplace([&](){

    tf::cudaGraph cg;

    // copy data to da, db, and dc
    auto copy_da = cg.copy(da, ha.data(), M*N);
    auto copy_db = cg.copy(db, hb.data(), N*K);
    auto copy_hc = cg.copy(hc.data(), dc, M*K); 

    dim3 grid  ((K+16-1)/16, (M+16-1)/16);
    dim3 block (16, 16);

    auto kmatmul = cg.kernel(grid, block, 0, matmul, da, db, dc, M, N, K);

    kmatmul.succeed(copy_da, copy_db)
           .precede(copy_hc);
    
    tf::cudaStream stream;
    tf::cudaGraphExec exec(cg);
    stream.run(exec).synchronize();
    cg.dump(std::cout);

  }).name("cudaFlow");

  auto free = taskflow.emplace([&](){
    TF_CHECK_CUDA(hipFree(da), "failed to free da");  
    TF_CHECK_CUDA(hipFree(db), "failed to free db");  
    TF_CHECK_CUDA(hipFree(dc), "failed to free dc");  
  }).name("free");

  cudaFlow.succeed(allocate_a, allocate_b, allocate_c)
          .precede(free);

  executor.run(taskflow).wait();
  
  // You may uncomment the line below to dump the task graph
  //taskflow.dump(std::cout);

  return hc;
}

// Matrix multiplication using CPU
auto cpu(int M, int N, int K) {  

  std::vector<int> a, b, c;

  tf::Executor executor;
  tf::Taskflow taskflow;

  auto ha = taskflow.emplace([&](){ 
    a.resize(M*N, M+N);
  }).name("allocate_a");

  auto hb = taskflow.emplace([&](){ 
    b.resize(N*K, N+K);
  }).name("allocate_b");

  auto hc = taskflow.emplace([&](){
    c.resize(M*K, 0);
  }).name("allocate_c");

  auto pf = taskflow.for_each_index(0, M, 1, [&] (int m) {
    for(int k=0; k<K; k++) {
      for(int n=0; n<N; n++) {
        c[m*K+k] += (a[m*N+n]*b[n*K+k]);
      }
    }
  });
  
  pf.succeed(ha, hb, hc);

  //taskflow.dump(std::cout);

  executor.run(taskflow).wait();

  return c;
}

// Function: main
int main(int argc, char *argv[]) {
  
  if(argc != 4) {
    std::cerr << "usage: matrix-multiplication M N K\n";
    std::exit(EXIT_FAILURE);
  }

  int M = std::atoi(argv[1]); 
  int N = std::atoi(argv[2]); 
  int K = std::atoi(argv[3]); 

  std::cout << "matrix A: " << M << 'x' << N << '\n'
            << "matrix B: " << N << 'x' << K << '\n'
            << "matrix C: " << M << 'x' << K << '\n';
  
  // matrix multiplication using gpu
  std::cout << "running gpu matrix multiplication ... ";
  auto gbeg = std::chrono::steady_clock::now();
  auto gres = gpu(M, N, K);
  auto gend = std::chrono::steady_clock::now();
  std::cout << "completed with " 
            << std::chrono::duration_cast<std::chrono::milliseconds>(gend-gbeg).count()
            << " ms\n";
  
  // matrix multiplication using cpu
  std::cout << "running cpu matrix multiplication ... ";
  auto cbeg = std::chrono::steady_clock::now();
  auto cres = cpu(M, N, K);
  auto cend = std::chrono::steady_clock::now();
  std::cout << "completed with " 
            << std::chrono::duration_cast<std::chrono::milliseconds>(cend-cbeg).count()
            << " ms\n";
  
  // verify the result
  int64_t error = 0;
  std::cout << "verifying results ... ";
  for(int i=0; i<M*K; ++i) {
    error += abs(gres[i] - cres[i]);
  }
  std::cout << "abs-error=" << error << '\n';

  return 0;
}










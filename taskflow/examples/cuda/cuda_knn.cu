#include "hip/hip_runtime.h"
// This program implements the k-means clustering algorithm in three forms:
//  - sequential cpu
//  - parallel cpu
//  - gpu with conditional tasking
//  - gpu without conditional tasking

#include <taskflow/taskflow.hpp>
#include <taskflow/algorithm/for_each.hpp>
#include <taskflow/cuda/cudaflow.hpp>

#include <iomanip>
#include <cfloat>
#include <climits>

#define L2(x1, y1, x2, y2) ((x1-x2)*(x1-x2) + (y1-y2)*(y1-y2))

// ----------------------------------------------------------------------------
// CPU (sequential) implementation
// ----------------------------------------------------------------------------

// run k-means on cpu
std::pair<std::vector<float>, std::vector<float>> cpu_seq(
 const int N, 
 const int K, 
 const int M,
 const std::vector<float>& px,
 const std::vector<float>& py
) {

  std::vector<int> c(K);
  std::vector<float> sx(K), sy(K), mx(K), my(K);
  
  // initial centroids
  for(int i=0; i<K; ++i) {
    mx[i] = px[i];
    my[i] = py[i];
  }
  
  for(int m=0; m<M; m++) {
  
    // clear the storage
    for(int k=0; k<K; ++k) {
      sx[k] = 0.0f;
      sy[k] = 0.0f;
      c [k] = 0;
    }

    // find the best k (cluster id) for each point
    for(int i=0; i<N; ++i) {
      float x = px[i];
      float y = py[i];
      float best_d = std::numeric_limits<float>::max();
      int best_k = 0;
      for (int k = 0; k < K; ++k) {
        const float d = L2(x, y, mx[k], my[k]);
        if (d < best_d) {
          best_d = d;
          best_k = k;
        }
      }
      sx[best_k] += x;
      sy[best_k] += y;
      c [best_k] += 1;
    }
    
    // update the centroid
    for(int k=0; k<K; k++) {
      const int count = max(1, c[k]);  // turn 0/0 to 0/1
      mx[k] = sx[k] / count;
      my[k] = sy[k] / count;
    }
  }
  
  return {mx, my};
}

// ----------------------------------------------------------------------------
// CPU (parallel) implementation
// ----------------------------------------------------------------------------

// run k-means on cpu (parallel)
std::pair<std::vector<float>, std::vector<float>> cpu_par(
 const int N, 
 const int K, 
 const int M,
 const std::vector<float>& px,
 const std::vector<float>& py
) {

  const auto num_threads = std::thread::hardware_concurrency();

  tf::Executor executor;
  tf::Taskflow taskflow("K-Means");
  
  std::vector<int> c(K), best_ks(N);
  std::vector<float> sx(K), sy(K), mx(K), my(K);
  
  // initial centroids
  auto init = taskflow.emplace([&](){
    for(int i=0; i<K; ++i) {
      mx[i] = px[i];
      my[i] = py[i];
    }
  }).name("init");
  
  // clear the storage
  auto clean_up = taskflow.emplace([&](){
    for(int k=0; k<K; ++k) {
      sx[k] = 0.0f;
      sy[k] = 0.0f;
      c [k] = 0;
    }
  }).name("clean_up");

  tf::Task pf;
  
  // update cluster
  pf = taskflow.for_each_index(0, N, 1, [&](int i){
    float x = px[i];
    float y = py[i];
    float best_d = std::numeric_limits<float>::max();
    int best_k = 0;
    for (int k = 0; k < K; ++k) {
      const float d = L2(x, y, mx[k], my[k]);
      if (d < best_d) {
        best_d = d;
        best_k = k;
      }
    }
    best_ks[i] = best_k;
  });

  pf.name("parallel-for");

  auto update_cluster = taskflow.emplace([&](){
    for(int i=0; i<N; i++) {
      sx[best_ks[i]] += px[i];
      sy[best_ks[i]] += py[i];
      c [best_ks[i]] += 1;
    }
    
    for(int k=0; k<K; ++k) {
      auto count = max(1, c[k]);  // turn 0/0 to 0/1
      mx[k] = sx[k] / count;
      my[k] = sy[k] / count;
    }
  }).name("update_cluster");

  auto condition = taskflow.emplace([m=0, M]() mutable {
    return (m++ < M) ? 0 : 1;
  }).name("converged?");
  
  init.precede(clean_up);

  clean_up.precede(pf);
  pf.precede(update_cluster);

  condition.precede(clean_up)
           .succeed(update_cluster);

  executor.run(taskflow).wait();
  
  return {mx, my};
}

// ----------------------------------------------------------------------------
// GPU implementation
// ----------------------------------------------------------------------------

// Each point (thread) computes its distance to each centroid 
// and adds its x and y values to the sum of its closest
// centroid, as well as incrementing that centroid's count of assigned points.
__global__ void assign_clusters(
  const float* px,
  const float* py,
  int N,
  const float* mx,
  const float* my,
  float* sx,
  float* sy,
  int k,
  int* c
) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= N) {
    return;
  }

  // Make global loads once.
  const float x = px[index];
  const float y = py[index];

  float best_distance = FLT_MAX;
  int best_cluster = 0;
  for (int cluster = 0; cluster < k; ++cluster) {
    const float distance = L2(x, y, mx[cluster], my[cluster]);
    if (distance < best_distance) {
      best_distance = distance;
      best_cluster = cluster;
    }
  }

  atomicAdd(&sx[best_cluster], x);
  atomicAdd(&sy[best_cluster], y);
  atomicAdd(&c [best_cluster], 1);
}

// Each thread is one cluster, which just recomputes its coordinates as the mean
// of all points assigned to it.
__global__ void compute_new_means(
  float* mx, float* my, const float* sx, const float* sy, const int* c
) {
  const int cluster = threadIdx.x;
  const int count = max(1, c[cluster]);  // turn 0/0 to 0/1
  mx[cluster] = sx[cluster] / count;
  my[cluster] = sy[cluster] / count;
}

// Runs k-means on gpu
std::pair<std::vector<float>, std::vector<float>> gpu_predicate(
 const int N, 
 const int K, 
 const int M,
 const std::vector<float>& h_px,
 const std::vector<float>& h_py
) {
  
  std::vector<float> h_mx, h_my;
  float *d_px, *d_py, *d_mx, *d_my, *d_sx, *d_sy, *d_c;
  
  for(int i=0; i<K; ++i) {
    h_mx.push_back(h_px[i]);
    h_my.push_back(h_py[i]);
  }
  
  // create a taskflow graph
  tf::Executor executor;
  tf::Taskflow taskflow("K-Means");
  
  auto allocate_px = taskflow.emplace([&](){
    TF_CHECK_CUDA(hipMalloc(&d_px, N*sizeof(float)), "failed to allocate d_px"); 
  }).name("allocate_px");

  auto allocate_py = taskflow.emplace([&](){
    TF_CHECK_CUDA(hipMalloc(&d_py, N*sizeof(float)), "failed to allocate d_py"); 
  }).name("allocate_py");
  
  auto allocate_mx = taskflow.emplace([&](){
    TF_CHECK_CUDA(hipMalloc(&d_mx, K*sizeof(float)), "failed to allocate d_mx"); 
  }).name("allocate_mx");

  auto allocate_my = taskflow.emplace([&](){
    TF_CHECK_CUDA(hipMalloc(&d_my, K*sizeof(float)), "failed to allocate d_my"); 
  }).name("allocate_my");

  auto allocate_sx = taskflow.emplace([&](){
    TF_CHECK_CUDA(hipMalloc(&d_sx, K*sizeof(float)), "failed to allocate d_sx"); 
  }).name("allocate_sx");

  auto allocate_sy = taskflow.emplace([&](){
    TF_CHECK_CUDA(hipMalloc(&d_sy, K*sizeof(float)), "failed to allocate d_sy"); 
  }).name("allocate_sy");

  auto allocate_c = taskflow.emplace([&](){
    TF_CHECK_CUDA(hipMalloc(&d_c, K*sizeof(float)), "failed to allocate dc");
  }).name("allocate_c");

  auto h2d = taskflow.emplace([&](){
    hipMemcpy(d_px, h_px.data(), N*sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_py, h_py.data(), N*sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_mx, h_mx.data(), K*sizeof(float), hipMemcpyDefault);
    hipMemcpy(d_my, h_my.data(), K*sizeof(float), hipMemcpyDefault);
  }).name("h2d");

  auto kmeans = taskflow.emplace([&](){

    tf::cudaGraph cg;

    auto zero_c = cg.zero(d_c, K);
    auto zero_sx = cg.zero(d_sx, K);
    auto zero_sy = cg.zero(d_sy, K);
    
    auto cluster = cg.kernel(
      (N+512-1) / 512, 512, 0, 
      assign_clusters, d_px, d_py, N, d_mx, d_my, d_sx, d_sy, K, d_c
    ); 
    
    auto new_centroid = cg.kernel(
      1, K, 0, 
      compute_new_means, d_mx, d_my, d_sx, d_sy, d_c
    );

    cluster.precede(new_centroid)
           .succeed(zero_c, zero_sx, zero_sy);
    
    // Repeat the execution for M times
    tf::cudaStream stream;
    tf::cudaGraphExec exec(cg);
    for(int i=0; i<M; i++) {
      stream.run(exec);
    }
    stream.synchronize();

    cg.dump(std::cout);
  }).name("update_means");

  auto stop = taskflow.emplace([&](){
    hipMemcpy(h_mx.data(), d_mx, K*sizeof(float), hipMemcpyDefault);
    hipMemcpy(h_my.data(), d_my, K*sizeof(float), hipMemcpyDefault);
  }).name("d2h");

  auto free = taskflow.emplace([&](){
    hipFree(d_px);
    hipFree(d_py);
    hipFree(d_mx);
    hipFree(d_my);
    hipFree(d_sx);
    hipFree(d_sy);
    hipFree(d_c);
  }).name("free");
  
  // build up the dependency
  h2d.succeed(allocate_px, allocate_py, allocate_mx, allocate_my);

  kmeans.succeed(allocate_sx, allocate_sy, allocate_c, h2d)
        .precede(stop);

  stop.precede(free);
  
  // run the taskflow
  executor.run(taskflow).wait();

  //std::cout << "dumping kmeans graph ...\n";
  taskflow.dump(std::cout);
  return {h_mx, h_my};
}

// Function: main
int main(int argc, const char* argv[]) {

  if(argc != 4) {
    std::cerr << "usage: ./kmeans num_points k num_iterations\n";
    std::exit(EXIT_FAILURE);
  }
  
  const int N = std::atoi(argv[1]);
  const int K = std::atoi(argv[2]);
  const int M = std::atoi(argv[3]);

  if(N < 1) {
    throw std::runtime_error("num_points must be at least one");
  }

  if(K >= N) {
    throw std::runtime_error("k must be smaller than the number of points");
  }

  if(M < 1) {
    throw std::runtime_error("num_iterations must be larger than 0");
  }

  std::vector<float> h_px, h_py, mx, my;
  
  // Randomly generate N points
  std::cout << "generating " << N << " random points ...\n";
  for(int i=0; i<N; ++i) {
    h_px.push_back(rand()%1000 - 500);
    h_py.push_back(rand()%1000 - 500);
  }

  // k-means on cpu_seq
  std::cout << "running k-means on cpu (sequential) ... ";
  auto sbeg = std::chrono::steady_clock::now();
  std::tie(mx, my) = cpu_seq(N, K, M, h_px, h_py);
  auto send = std::chrono::steady_clock::now();
  std::cout << "completed with " 
            << std::chrono::duration_cast<std::chrono::milliseconds>(send-sbeg).count()
            << " ms\n";
  
  std::cout << "k centroids found by cpu (sequential)\n";
  for(int k=0; k<K; ++k) {
    std::cout << "centroid " << k << ": " << std::setw(10) << mx[k] << ' ' 
                                          << std::setw(10) << my[k] << '\n';  
  }
  
  // k-means on cpu_par
  std::cout << "running k-means on cpu (parallel) ... ";
  auto pbeg = std::chrono::steady_clock::now();
  std::tie(mx, my) = cpu_par(N, K, M, h_px, h_py);
  auto pend = std::chrono::steady_clock::now();
  std::cout << "completed with " 
            << std::chrono::duration_cast<std::chrono::milliseconds>(pend-pbeg).count()
            << " ms\n";
  
  std::cout << "k centroids found by cpu (parallel)\n";
  for(int k=0; k<K; ++k) {
    std::cout << "centroid " << k << ": " << std::setw(10) << mx[k] << ' ' 
                                          << std::setw(10) << my[k] << '\n';  
  }
  
  // k-means on gpu 
  std::cout << "running k-means on gpu ...";
  auto rbeg = std::chrono::steady_clock::now();
  std::tie(mx, my) = gpu_predicate(N, K, M, h_px, h_py);
  auto rend = std::chrono::steady_clock::now();
  std::cout << "completed with " 
            << std::chrono::duration_cast<std::chrono::milliseconds>(rend-rbeg).count()
            << " ms\n";
  
  std::cout << "k centroids found by gpu\n";
  for(int k=0; k<K; ++k) {
    std::cout << "centroid " << k << ": " << std::setw(10) << mx[k] << ' ' 
                                          << std::setw(10) << my[k] << '\n';  
  }

  return 0;
}




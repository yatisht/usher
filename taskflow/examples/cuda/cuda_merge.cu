#include <taskflow/cuda/cudaflow.hpp>
#include <taskflow/cuda/algorithm/merge.hpp>

int main(int argc, char* argv[]) {
  
  if(argc != 2) {
    std::cerr << "usage: ./cuda_merge N\n";
    std::exit(EXIT_FAILURE);
  }

  unsigned N = std::atoi(argv[1]);
  
  // gpu data
  auto da = tf::cuda_malloc_shared<int>(N);
  auto db = tf::cuda_malloc_shared<int>(N);
  auto dc = tf::cuda_malloc_shared<int>(N + N);

  // host data
  std::vector<int> ha(N), hb(N), hc(N + N);

  for(unsigned i=0; i<N; i++) {
    da[i] = ha[i] = rand()%100;
    db[i] = hb[i] = rand()%100;
  }
  
  std::sort(da, da+N);
  std::sort(db, db+N);
  std::sort(ha.begin(), ha.end());
  std::sort(hb.begin(), hb.end());

  // --------------------------------------------------------------------------
  // GPU merge
  // --------------------------------------------------------------------------

  tf::cudaStream stream;
  tf::cudaDefaultExecutionPolicy policy(stream);

  // allocate the buffer
  void* buf;
  hipMalloc(&buf, policy.merge_bufsz(N, N));

  auto beg = std::chrono::steady_clock::now();
  tf::cuda_merge(policy, 
    da, da+N, db, db+N, dc, tf::cuda_less<int>{}, buf
  );
  stream.synchronize();
  auto end = std::chrono::steady_clock::now();

  std::cout << "GPU merge: " 
            << std::chrono::duration_cast<std::chrono::microseconds>(end-beg).count()
            << " us\n";
  
  // --------------------------------------------------------------------------
  // CPU merge
  // --------------------------------------------------------------------------
  beg = std::chrono::steady_clock::now();
  std::merge(ha.begin(), ha.end(), hb.begin(), hb.end(), hc.begin());
  end = std::chrono::steady_clock::now();
  
  std::cout << "CPU merge: " 
            << std::chrono::duration_cast<std::chrono::microseconds>(end-beg).count()
            << " us\n";

  // --------------------------------------------------------------------------
  // verify the result
  // --------------------------------------------------------------------------

  for(size_t i=0; i<N; i++) {
    if(dc[i] != hc[i]) {
      throw std::runtime_error("incorrect result");
    }
  }

  std::cout << "correct result\n";
  
  // --------------------------------------------------------------------------
  // deallocate the memory
  // --------------------------------------------------------------------------
  hipFree(da);
  hipFree(db);
  hipFree(dc);
  hipFree(buf);

  return 0;
};

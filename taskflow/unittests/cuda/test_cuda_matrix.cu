#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>
#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cudaflow.hpp>

// ----------------------------------------------------------------------------
// Matrix Multiplication Kernel
// ----------------------------------------------------------------------------
__global__ void k_multiplication(
  int *a, int *b, int *c, int m, int n, int k
) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int sum = 0;
  if(col < k && row < m) {
    for(int i = 0; i < n; i++) {
      sum += a[row * n + i] * b[i * k + col];
    }
    c[row * k + col] = sum;
  }
}

TEST_CASE("multiply" * doctest::timeout(300)) {

  tf::Taskflow taskflow;
  tf::Executor executor;
  
  std::vector<int> a, b, c;

  const unsigned B = 16;

  for(int m=1; m<=256; m<<=1) {
    for(int n=1; n<=256; n<<=1) {
      for(int k=1; k<=256; k<<=1) {

        taskflow.clear();

        int* ha {nullptr};
        int* hb {nullptr};
        int* hc {nullptr};
        int* da {nullptr};
        int* db {nullptr};
        int* dc {nullptr};

        dim3 grid  ((k+B-1)/B, (m+B-1)/B);
        dim3 block (B, B);

        auto hosta = taskflow.emplace([&](){ 
          a.resize(m*n);
          std::fill_n(a.begin(), m*n, m+n);
          ha = a.data();
          REQUIRE(hipMalloc(&da, m*n*sizeof(int)) == hipSuccess);
        }).name("ha");

        auto hostb = taskflow.emplace([&](){ 
          b.resize(n*k);
          std::fill_n(b.begin(), n*k, n+k);
          hb = b.data();
          REQUIRE(hipMalloc(&db, n*k*sizeof(int)) == hipSuccess);
        }).name("hb");

        auto hostc = taskflow.emplace([&](){
          c.resize(m*k);
          hc = c.data();
          REQUIRE(hipMalloc(&dc, m*k*sizeof(int)) == hipSuccess);
        }).name("hc");

        auto cuda = taskflow.emplace([&](){
          tf::cudaGraph cg;
          auto pa = cg.copy(da, ha, m*n);
          auto pb = cg.copy(db, hb, n*k);

          auto op = cg.kernel(
            grid, block, 0, k_multiplication, da, db, dc, m, n, k
          );

          auto cc = cg.copy(hc, dc, m*k);

          op.precede(cc).succeed(pa, pb);

          tf::cudaGraphExec exec(cg); 
          tf::cudaStream stream;
          stream.run(exec)
                .synchronize();
        });

        cuda.succeed(hosta, hostb, hostc);

        executor.run(taskflow).wait();

        for(const auto& x : c) {
          REQUIRE(x == (int)(m+n)*(n+k)*n);
        }

        REQUIRE(hipFree(da) == hipSuccess);
        REQUIRE(hipFree(db) == hipSuccess);
        REQUIRE(hipFree(dc) == hipSuccess);
      }
    }
  }
}

// ----------------------------------------------------------------------------
// Matrix Transpose
// ----------------------------------------------------------------------------
__global__ void k_transpose(int *mat_in, int *mat_out, int rows, int cols) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
  if (idx < cols && idy < rows) {
    unsigned int pos = idy * cols + idx;
    unsigned int trans_pos = idx * rows + idy;
    mat_out[trans_pos] = mat_in[pos];
  }
}

TEST_CASE("transpose" * doctest::timeout(300)) {
  
  std::vector<int> in, out;
  
  tf::Taskflow taskflow;
  tf::Executor executor;

  const unsigned B = 16;

  for(int m=1; m<=256; m<<=1) {
    for(int n=1; n<=256; n<<=1) {

      taskflow.clear();

      int* ptr_in {nullptr};
      int* ptr_out {nullptr};
      int* sin {nullptr};
      int* sout {nullptr};

      dim3 grid  ((n+B-1)/B, (m+B-1)/B);
      dim3 block (B, B);

      auto hin = taskflow.emplace([&](){ 
        in.resize(m*n);
        out.resize(m*n);
        for(auto& item : in) {
          item = ::rand()%100;
        }
        ptr_in = in.data();
        ptr_out = out.data();
        REQUIRE(hipMalloc(&sin, m*n*sizeof(int)) == hipSuccess);
        REQUIRE(hipMalloc(&sout, m*n*sizeof(int)) == hipSuccess);
      }).name("ha");

      auto op = taskflow.emplace([&](){
        tf::cudaGraph cg;
        auto copyin = cg.copy(sin, ptr_in, m*n);
        auto copyout = cg.copy(ptr_out, sout, m*n);
        auto trans = cg.kernel(grid, block, 0, k_transpose, sin, sout, m, n);
        trans.succeed(copyin).precede(copyout);
        tf::cudaGraphExec exec(cg);
        tf::cudaStream stream;
        stream.run(exec).synchronize();
      });

      hin.precede(op);

      executor.run(taskflow).wait();

      for(int x=0; x<m; x++) {
        for(int y=0; y<n; ++y) {
          REQUIRE(in[x*n+y] == out[y*m+x]);
        }
      }

      REQUIRE(hipFree(sin) == hipSuccess);
      REQUIRE(hipFree(sout) == hipSuccess);
    }
  }
}

// ----------------------------------------------------------------------------
// vector product
// ----------------------------------------------------------------------------
__global__ void k_product(int *a, int *b, int *c, int N) {
  unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    c[idx] = a[idx] * b[idx];
  }
}

TEST_CASE("product" * doctest::timeout(300)) {

  tf::Taskflow taskflow;
  tf::Executor executor;

  const unsigned num_batches = 1024;
  const unsigned N = 1024;
  const unsigned B = 128;

  dim3 grid  ((N+B-1)/B);
  dim3 block (B);

  std::vector<int*> hA(num_batches);
  std::vector<int*> hB(num_batches);
  std::vector<int*> hC(num_batches);
  std::vector<int*> dA(num_batches);
  std::vector<int*> dB(num_batches);
  std::vector<int*> dC(num_batches);

  for(unsigned i=0; i<num_batches; ++i) {

    int v1 = ::rand()%10;
    int v2 = ::rand()%10;

    auto allocate = taskflow.emplace([&, i, v1, v2](){
      hA[i] = new int [N];
      hB[i] = new int [N];
      hC[i] = new int [N];
      REQUIRE(hipMalloc(&dA[i], N*sizeof(int)) == hipSuccess);
      REQUIRE(hipMalloc(&dB[i], N*sizeof(int)) == hipSuccess);
      REQUIRE(hipMalloc(&dC[i], N*sizeof(int)) == hipSuccess);
      for(unsigned j=0; j<N; ++j) {
        hA[i][j] = v1;
        hB[i][j] = v2;
      }
    });

    auto kernel = taskflow.emplace([&, i](){
      tf::cudaGraph cg;
      auto copyA = cg.copy(dA[i], hA[i], N);
      auto copyB = cg.copy(dB[i], hB[i], N);
      auto op = cg.kernel(grid, block, 0, k_product, dA[i], dB[i], dC[i], N);
      auto copyC = cg.copy(hC[i], dC[i], N);
      op.succeed(copyA, copyB).precede(copyC);
      tf::cudaStream stream;
      tf::cudaGraphExec exec(cg);
      stream.run(exec)
            .synchronize();
    });

    auto deallocate = taskflow.emplace([&, i, v1, v2](){
      for(unsigned j=0; j<N; ++j) {
        REQUIRE(hC[i][j] == v1*v2);
      }
      delete hA[i];
      delete hB[i];
      delete hC[i];
      REQUIRE(hipFree(dA[i]) == hipSuccess);
      REQUIRE(hipFree(dB[i]) == hipSuccess);
      REQUIRE(hipFree(dC[i]) == hipSuccess);
    });

    kernel.precede(deallocate).succeed(allocate);
  }

  executor.run(taskflow).wait();

}










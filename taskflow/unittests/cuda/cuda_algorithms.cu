#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>
#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cudaflow.hpp>
#include <taskflow/cuda/algorithm/for_each.hpp>
#include <taskflow/cuda/algorithm/transform.hpp>
#include <taskflow/cuda/algorithm/reduce.hpp>
#include <taskflow/cuda/algorithm/sort.hpp>
#include <taskflow/cuda/algorithm/find.hpp>
#include <taskflow/cuda/algorithm/scan.hpp>

constexpr float eps = 0.0001f;

template <typename T>
void run_and_wait(T& cf) {
  tf::cudaStream stream;
  cf.run(stream);
  stream.synchronize();
}

// --------------------------------------------------------
// Testcase: add2
// --------------------------------------------------------
template <typename T, typename F>
void add2() {

  //const unsigned N = 1<<20;
    
  tf::Taskflow taskflow;
  tf::Executor executor;

  for(size_t N=1; N<=(1<<20); N <<= 1) {

    taskflow.clear();

    T v1 = ::rand() % 100;
    T v2 = ::rand() % 100;

    std::vector<T> hx, hy;

    T* dx {nullptr};
    T* dy {nullptr};
    
    // allocate x
    auto allocate_x = taskflow.emplace([&]() {
      hx.resize(N, v1);
      REQUIRE(hipMalloc(&dx, N*sizeof(T)) == hipSuccess);
    }).name("allocate_x");

    // allocate y
    auto allocate_y = taskflow.emplace([&]() {
      hy.resize(N, v2);
      REQUIRE(hipMalloc(&dy, N*sizeof(T)) == hipSuccess);
    }).name("allocate_y");
    
    // axpy
    auto cudaflow = taskflow.emplace([&]() {
      F cf;
      auto h2d_x = cf.copy(dx, hx.data(), N).name("h2d_x");
      auto h2d_y = cf.copy(dy, hy.data(), N).name("h2d_y");
      auto d2h_x = cf.copy(hx.data(), dx, N).name("d2h_x");
      auto d2h_y = cf.copy(hy.data(), dy, N).name("d2h_y");
      //auto kernel = cf.add(dx, N, dx, dy);
      auto kernel = cf.transform(dx, dx+N, dy, 
        [] __device__ (T x) { return x + 2;  }
      );
      kernel.succeed(h2d_x, h2d_y)
            .precede(d2h_x, d2h_y);

      run_and_wait(cf);
    }).name("saxpy");

    cudaflow.succeed(allocate_x, allocate_y);

    // Add a verification task
    auto verifier = taskflow.emplace([&](){
      for (size_t i = 0; i < N; i++) {
        REQUIRE(std::fabs(hx[i] - v1) < eps);
        REQUIRE(std::fabs(hy[i] - (hx[i] + 2)) < eps);
      }
    }).succeed(cudaflow).name("verify");

    // free memory
    auto deallocate_x = taskflow.emplace([&](){
      REQUIRE(hipFree(dx) == hipSuccess);
    }).name("deallocate_x");
    
    auto deallocate_y = taskflow.emplace([&](){
      REQUIRE(hipFree(dy) == hipSuccess);
    }).name("deallocate_y");

    verifier.precede(deallocate_x, deallocate_y);

    executor.run(taskflow).wait();

    // standalone tramsform
    tf::cudaDefaultExecutionPolicy p;

    auto input  = tf::cuda_malloc_shared<T>(N);
    auto output = tf::cuda_malloc_shared<T>(N);
    for(size_t n=0; n<N; n++) {
      input[n] = 1;
    }

    tf::cuda_transform(p, input, input + N, output, 
      [] __device__ (T i) { return i+2; }
    );
    hipStreamSynchronize(0);

    for(size_t n=0; n<N; n++) {
      REQUIRE(output[n] == 3);
    }
  }
}

TEST_CASE("add2.int" * doctest::timeout(300)) {
  add2<int, tf::cudaFlow>();
}

TEST_CASE("add2.float" * doctest::timeout(300)) {
  add2<float, tf::cudaFlow>();
}

TEST_CASE("add2.double" * doctest::timeout(300)) {
  add2<double, tf::cudaFlow>();
}

TEST_CASE("capture_add2.int" * doctest::timeout(300)) {
  add2<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capture_add2.float" * doctest::timeout(300)) {
  add2<float, tf::cudaFlowCapturer>();
}

TEST_CASE("capture_add2.double" * doctest::timeout(300)) {
  add2<double, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// transform
// ----------------------------------------------------------------------------

template <typename F>
void transform() {
  
  F cudaflow;
    
  for(unsigned n=1; n<=1234567; n = n*2 + 1) {

    cudaflow.clear();

    auto src1 = tf::cuda_malloc_shared<int>(n);
    auto src2 = tf::cuda_malloc_shared<int>(n);
    auto dest = tf::cuda_malloc_shared<int>(n);

    for(unsigned i=0; i<n; i++) {
      src1[i] = 10;
      src2[i] = 90;
      dest[i] = 0;
    }

    cudaflow.transform(src1, src1+n, src2, dest,
      []__device__(int s1, int s2) { return s1 + s2; } 
    );

    run_and_wait(cudaflow);

    for(unsigned i=0; i<n; i++){
      REQUIRE(dest[i] == src1[i] + src2[i]);
    }
  }
}

TEST_CASE("cudaflow.transform" * doctest::timeout(300)) {
  transform<tf::cudaFlow>();
}

TEST_CASE("capture.transform" * doctest::timeout(300) ) {
  transform<tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// reduce
// ----------------------------------------------------------------------------

template <typename T, typename F>
void reduce() {

  for(int n=1; n<=1234567; n = n*2 + 1) {

    tf::Taskflow taskflow;
    tf::Executor executor;

    T sum = 0;

    std::vector<T> cpu(n);
    for(auto& i : cpu) {
      i = ::rand()%100-50;
      sum += i;
    }

    T sol;
    
    T* gpu = nullptr;
    T* res = nullptr;

    auto cputask = taskflow.emplace([&](){
      REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
      REQUIRE(hipMalloc(&res, 1*sizeof(T)) == hipSuccess);
    });

    tf::Task gputask;
    
    gputask = taskflow.emplace([&]() {
      F cf;
      auto d2h = cf.copy(&sol, res, 1);
      auto h2d = cf.copy(gpu, cpu.data(), n);
      auto set = cf.single_task([res] __device__ () mutable {
        *res = 1000;
      });
      auto kernel = cf.reduce(
        gpu, gpu+n, res, [] __device__ (T a, T b) mutable { 
          return a + b;
        }
      );
      kernel.succeed(h2d, set);
      d2h.succeed(kernel);
      run_and_wait(cf);
    });

    cputask.precede(gputask);
    
    executor.run(taskflow).wait();

    REQUIRE(std::fabs(sum-sol+1000) < 0.0001);

    REQUIRE(hipFree(gpu) == hipSuccess);
    REQUIRE(hipFree(res) == hipSuccess);
  }
}

TEST_CASE("cudaflow.reduce.int" * doctest::timeout(300)) {
  reduce<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.reduce.float" * doctest::timeout(300)) {
  reduce<float, tf::cudaFlow>();
}

TEST_CASE("cudaflow.reduce.double" * doctest::timeout(300)) {
  reduce<double, tf::cudaFlow>();
}

TEST_CASE("capture.reduce.int" * doctest::timeout(300)) {
  reduce<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.reduce.float" * doctest::timeout(300)) {
  reduce<float, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.reduce.double" * doctest::timeout(300)) {
  reduce<double, tf::cudaFlow>();
}

// ----------------------------------------------------------------------------
// uninitialized_reduce
// ----------------------------------------------------------------------------

template <typename T, typename F>
void uninitialized_reduce() {

  for(int n=1; n<=1234567; n = n*2 + 1) {

    tf::Taskflow taskflow;
    tf::Executor executor;

    T sum = 0;

    std::vector<T> cpu(n);
    for(auto& i : cpu) {
      i = ::rand()%100-50;
      sum += i;
    }

    T sol;
    
    T* gpu = nullptr;
    T* res = nullptr;

    auto cputask = taskflow.emplace([&](){
      REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
      REQUIRE(hipMalloc(&res, 1*sizeof(T)) == hipSuccess);
    });

    tf::Task gputask;
    
    gputask = taskflow.emplace([&]() {
      F cf;
      auto d2h = cf.copy(&sol, res, 1);
      auto h2d = cf.copy(gpu, cpu.data(), n);
      auto set = cf.single_task([res] __device__ () mutable {
        *res = 1000;
      });
      auto kernel = cf.uninitialized_reduce(
        gpu, gpu+n, res, [] __device__ (T a, T b) { 
          return a + b;
        }
      );
      kernel.succeed(h2d, set);
      d2h.succeed(kernel);
      run_and_wait(cf);
    });

    cputask.precede(gputask);
    
    executor.run(taskflow).wait();

    REQUIRE(std::fabs(sum-sol) < 0.0001);

    REQUIRE(hipFree(gpu) == hipSuccess);
    REQUIRE(hipFree(res) == hipSuccess);
  }
}

TEST_CASE("cudaflow.uninitialized_reduce.int" * doctest::timeout(300)) {
  uninitialized_reduce<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.uninitialized_reduce.float" * doctest::timeout(300)) {
  uninitialized_reduce<float, tf::cudaFlow>();
}

TEST_CASE("cudaflow.uninitialized_reduce.double" * doctest::timeout(300)) {
  uninitialized_reduce<double, tf::cudaFlow>();
}

TEST_CASE("capture.uninitialized_reduce.int" * doctest::timeout(300)) {
  uninitialized_reduce<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.uninitialized_reduce.float" * doctest::timeout(300)) {
  uninitialized_reduce<float, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.uninitialized_reduce.double" * doctest::timeout(300)) {
  uninitialized_reduce<double, tf::cudaFlow>();
}

// ----------------------------------------------------------------------------
// transform_reduce
// ----------------------------------------------------------------------------

template <typename T, typename F>
void transform_reduce() {
    
  tf::Executor executor;

  for(int n=1; n<=1234567; n = n*2 + 1) {

    tf::Taskflow taskflow;

    T sum = 0;

    std::vector<T> cpu(n);
    for(auto& i : cpu) {
      i = ::rand()%100-50;
      sum += i;
    }

    T sol;
    
    T* gpu = nullptr;
    T* res = nullptr;

    auto cputask = taskflow.emplace([&](){
      REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
      REQUIRE(hipMalloc(&res, 1*sizeof(T)) == hipSuccess);
    });

    tf::Task gputask;
    
    gputask = taskflow.emplace([&]() {
      F cf;
      auto d2h = cf.copy(&sol, res, 1);
      auto h2d = cf.copy(gpu, cpu.data(), n);
      auto set = cf.single_task([res] __device__ () mutable {
        *res = 1000;
      });
      auto kernel = cf.transform_reduce(
        gpu, gpu+n, res, 
        [] __device__ (T a, T b) { return a + b; },
        [] __device__ (T a) { return a + 1; }
      );
      kernel.succeed(h2d, set);
      d2h.succeed(kernel);
      run_and_wait(cf);
    });

    cputask.precede(gputask);
    
    executor.run(taskflow).wait();

    REQUIRE(std::fabs(sum+n+1000-sol) < 0.0001);

    REQUIRE(hipFree(gpu) == hipSuccess);
    REQUIRE(hipFree(res) == hipSuccess);
  }
}

TEST_CASE("cudaflow.transform_reduce.int" * doctest::timeout(300)) {
  transform_reduce<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.transform_reduce.float" * doctest::timeout(300)) {
  transform_reduce<float, tf::cudaFlow>();
}

TEST_CASE("cudaflow.transform_reduce.double" * doctest::timeout(300)) {
  transform_reduce<double, tf::cudaFlow>();
}

TEST_CASE("capture.transform_reduce.int" * doctest::timeout(300)) {
  transform_reduce<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.transform_reduce.float" * doctest::timeout(300)) {
  transform_reduce<float, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.transform_reduce.double" * doctest::timeout(300)) {
  transform_reduce<double, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// transform_uninitialized_reduce
// ----------------------------------------------------------------------------

template <typename T, typename F>
void transform_uninitialized_reduce() {
    
  tf::Executor executor;

  for(int n=1; n<=1234567; n = n*2 + 1) {

    tf::Taskflow taskflow;

    T sum = 0;

    std::vector<T> cpu(n);
    for(auto& i : cpu) {
      i = ::rand()%100-50;
      sum += i;
    }

    T sol;
    
    T* gpu = nullptr;
    T* res = nullptr;

    auto cputask = taskflow.emplace([&](){
      REQUIRE(hipMalloc(&gpu, n*sizeof(T)) == hipSuccess);
      REQUIRE(hipMalloc(&res, 1*sizeof(T)) == hipSuccess);
    });

    tf::Task gputask;
    
    gputask = taskflow.emplace([&]() {
      F cf;
      auto d2h = cf.copy(&sol, res, 1);
      auto h2d = cf.copy(gpu, cpu.data(), n);
      auto set = cf.single_task([res] __device__ () mutable {
        *res = 1000;
      });
      auto kernel = cf.transform_uninitialized_reduce(
        gpu, gpu+n, res, 
        [] __device__ (T a, T b) { return a + b; },
        [] __device__ (T a) { return a + 1; }
      );
      kernel.succeed(h2d, set);
      d2h.succeed(kernel);
      run_and_wait(cf);
    });

    cputask.precede(gputask);
    
    executor.run(taskflow).wait();

    REQUIRE(std::fabs(sum+n-sol) < 0.0001);

    REQUIRE(hipFree(gpu) == hipSuccess);
    REQUIRE(hipFree(res) == hipSuccess);
  }
}

TEST_CASE("cudaflow.transform_uninitialized_reduce.int" * doctest::timeout(300)) {
  transform_uninitialized_reduce<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.transform_uninitialized_reduce.float" * doctest::timeout(300)) {
  transform_uninitialized_reduce<float, tf::cudaFlow>();
}

TEST_CASE("cudaflow.transform_uninitialized_reduce.double" * doctest::timeout(300)) {
  transform_uninitialized_reduce<double, tf::cudaFlow>();
}

TEST_CASE("capture.transform_uninitialized_reduce.int" * doctest::timeout(300)) {
  transform_uninitialized_reduce<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.transform_uninitialized_reduce.float" * doctest::timeout(300)) {
  transform_uninitialized_reduce<float, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.transform_uninitialized_reduce.double" * doctest::timeout(300)) {
  transform_uninitialized_reduce<double, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// scan
// ----------------------------------------------------------------------------

template <typename T, typename F>
void scan() {
    
  tf::Executor executor;
  tf::Taskflow taskflow;

  for(int N=1; N<=1234567; N = N*2 + 1) {

    taskflow.clear();
  
    auto data1 = tf::cuda_malloc_shared<T>(N);
    auto data2 = tf::cuda_malloc_shared<T>(N);
    auto scan1 = tf::cuda_malloc_shared<T>(N);
    auto scan2 = tf::cuda_malloc_shared<T>(N);

    // initialize the data
    for(int i=0; i<N; i++) {
      data1[i] = T(i);
      data2[i] = T(i);
      scan1[i] = 0;
      scan2[i] = 0;
    }
    
    // perform reduction
    taskflow.emplace([&](){
      F cudaflow;
      // inclusive scan
      cudaflow.inclusive_scan(
        data1, data1+N, scan1, [] __device__ (T a, T b){ return a+b; }
      );
      // exclusive scan
      cudaflow.exclusive_scan(
        data2, data2+N, scan2, [] __device__ (T a, T b){ return a+b; }
      );
      run_and_wait(cudaflow);
    });

    executor.run(taskflow).wait();
    
    // inspect 
    for(int i=1; i<N; i++) {
      REQUIRE(scan1[i] == (scan1[i-1]+data1[i]));
      REQUIRE(scan2[i] == (scan2[i-1]+data2[i-1]));
    }

    // test standalone algorithms
    
    // initialize the data
    for(int i=0; i<N; i++) {
      data1[i] = T(i);
      data2[i] = T(i);
      scan1[i] = 0;
      scan2[i] = 0;
    }

    // allocate temporary buffer
    tf::cudaDeviceVector<std::byte> temp(
      tf::cuda_scan_buffer_size<tf::cudaDefaultExecutionPolicy, T>(N)
    );
      
    tf::cuda_inclusive_scan(tf::cudaDefaultExecutionPolicy{}, 
      data1, data1+N, scan1, tf::cuda_plus<T>{}, temp.data()
    );
    hipStreamSynchronize(0);

    tf::cuda_exclusive_scan(tf::cudaDefaultExecutionPolicy{}, 
      data2, data2+N, scan2, tf::cuda_plus<T>{}, temp.data()
    );
    hipStreamSynchronize(0);
    
    // inspect 
    for(int i=1; i<N; i++) {
      REQUIRE(scan1[i] == (scan1[i-1]+data1[i]));
      REQUIRE(scan2[i] == (scan2[i-1]+data2[i-1]));
    }

    REQUIRE(hipFree(data1) == hipSuccess);
    REQUIRE(hipFree(data2) == hipSuccess);
    REQUIRE(hipFree(scan1) == hipSuccess);
    REQUIRE(hipFree(scan2) == hipSuccess);
  }
}

TEST_CASE("cudaflow.scan.int" * doctest::timeout(300)) {
  scan<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.scan.size_t" * doctest::timeout(300)) {
  scan<size_t, tf::cudaFlow>();
}

TEST_CASE("capture.scan.int" * doctest::timeout(300)) {
  scan<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.scan.size_t" * doctest::timeout(300)) {
  scan<size_t, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// transofrm scan
// ----------------------------------------------------------------------------

template <typename T, typename F>
void transform_scan() {
    
  tf::Executor executor;
  tf::Taskflow taskflow;

  for(int N=1; N<=1234567; N = N*2 + 1) {

    taskflow.clear();
  
    auto data1 = tf::cuda_malloc_shared<T>(N);
    auto data2 = tf::cuda_malloc_shared<T>(N);
    auto scan1 = tf::cuda_malloc_shared<T>(N);
    auto scan2 = tf::cuda_malloc_shared<T>(N);

    // initialize the data
    for(int i=0; i<N; i++) {
      data1[i] = T(i);
      data2[i] = T(i);
      scan1[i] = 0;
      scan2[i] = 0;
    }
    
    // perform reduction
    taskflow.emplace([&](){

      F cudaflow;

      // inclusive scan
      cudaflow.transform_inclusive_scan(
        data1, data1+N, scan1, 
        [] __device__ (T a, T b){ return a+b; },
        [] __device__ (T a) { return a*10; }
      );
      // exclusive scan
      cudaflow.transform_exclusive_scan(
        data2, data2+N, scan2, 
        [] __device__ (T a, T b){ return a+b; },
        [] __device__ (T a) { return a*10; }
      );

      run_and_wait(cudaflow);
    });

    executor.run(taskflow).wait();
    
    // standalone algorithms
    
    // initialize the data
    for(int i=0; i<N; i++) {
      data1[i] = T(i);
      data2[i] = T(i);
      scan1[i] = 0;
      scan2[i] = 0;
    }
    
    // allocate temporary buffer
    tf::cudaDeviceVector<std::byte> temp(
      tf::cuda_scan_buffer_size<tf::cudaDefaultExecutionPolicy, T>(N)
    );
      
    tf::cuda_transform_inclusive_scan(tf::cudaDefaultExecutionPolicy{},
      data1, data1+N, scan1, 
      [] __device__ (T a, T b){ return a+b; },
      [] __device__ (T a) { return a*10; },
      temp.data()
    );
    hipStreamSynchronize(0);
      
    tf::cuda_transform_exclusive_scan(tf::cudaDefaultExecutionPolicy{},
      data2, data2+N, scan2, 
      [] __device__ (T a, T b){ return a+b; },
      [] __device__ (T a) { return a*10; },
      temp.data()
    );
    hipStreamSynchronize(0);
    
    // inspect 
    for(int i=1; i<N; i++) {
      REQUIRE(scan1[i] == (scan1[i-1]+data1[i]*10));
      REQUIRE(scan2[i] == (scan2[i-1]+data2[i-1]*10));
    }

    REQUIRE(hipFree(data1) == hipSuccess);
    REQUIRE(hipFree(data2) == hipSuccess);
    REQUIRE(hipFree(scan1) == hipSuccess);
    REQUIRE(hipFree(scan2) == hipSuccess);
  }
}

TEST_CASE("cudaflow.scan.int" * doctest::timeout(300)) {
  transform_scan<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.scan.size_t" * doctest::timeout(300)) {
  transform_scan<size_t, tf::cudaFlow>();
}

TEST_CASE("capture.transform_scan.int" * doctest::timeout(300)) {
  transform_scan<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.transform_scan.size_t" * doctest::timeout(300)) {
  transform_scan<size_t, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// merge
// ----------------------------------------------------------------------------

template <typename T, typename F>
void merge_keys() {
    
  tf::Executor executor;
  tf::Taskflow taskflow;

  for(int N=0; N<=1234567; N = N*2 + 1) {

    taskflow.clear();

    auto a = tf::cuda_malloc_shared<T>(N);
    auto b = tf::cuda_malloc_shared<T>(N);
    auto c = tf::cuda_malloc_shared<T>(2*N);
  
    tf::cudaStream s;
    auto p = tf::cudaDefaultExecutionPolicy{s};

    // ----------------- standalone algorithms

    // initialize the data
    for(int i=0; i<N; i++) {
      a[i] = T(rand()%100);
      b[i] = T(rand()%100);
    }

    std::sort(a, a+N);
    std::sort(b, b+N);
    
    auto bufsz = tf::cuda_merge_buffer_size<decltype(p)>(N, N);
    tf::cudaDeviceVector<std::byte> buf(bufsz);

    tf::cuda_merge(p, a, a+N, b, b+N, c, tf::cuda_less<T>{}, buf.data());
    s.synchronize();

    REQUIRE(std::is_sorted(c, c+2*N));
    
    /*// ----------------- cudaFlow capturer
    for(int i=0; i<N*2; i++) {
      c[i] = rand();      
    }
    
    taskflow.emplace([&](){
      F cudaflow;
      cudaflow.merge(a, a+N, b, b+N, c, tf::cuda_less<T>{});
      run_and_wait(cudaflow);
    });

    executor.run(taskflow).wait();
    
    REQUIRE(std::is_sorted(c, c+2*N));*/
    
    REQUIRE(hipFree(a) == hipSuccess);
    REQUIRE(hipFree(b) == hipSuccess);
    REQUIRE(hipFree(c) == hipSuccess);
  }
}

TEST_CASE("cudaflow.merge_keys.int" * doctest::timeout(300)) {
  merge_keys<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.merge_keys.float" * doctest::timeout(300)) {
  merge_keys<float, tf::cudaFlow>();
}

TEST_CASE("cudaflow.merge_keys.int" * doctest::timeout(300)) {
  merge_keys<int, tf::cudaFlow>();
}

TEST_CASE("capture.merge_keys.float" * doctest::timeout(300)) {
  merge_keys<float, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// merge_by_keys
// ----------------------------------------------------------------------------

template <typename T, typename F>
void merge_keys_values() {
    
  tf::Executor executor;
  tf::Taskflow taskflow;

  for(int N=0; N<=1234567; N = N*2 + 1) {

    taskflow.clear();

    auto a_k = tf::cuda_malloc_shared<T>(N);
    auto b_k = tf::cuda_malloc_shared<T>(N);
    auto c_k = tf::cuda_malloc_shared<T>(2*N);
    auto a_v = tf::cuda_malloc_shared<int>(N);
    auto b_v = tf::cuda_malloc_shared<int>(N);
    auto c_v = tf::cuda_malloc_shared<int>(2*N);

    tf::cudaStream s;
    auto p = tf::cudaDefaultExecutionPolicy{s};

    // ----------------- standalone algorithms

    // initialize the data
    for(int i=0; i<N; i++) {
      a_k[i] =  (i*2+1);
      a_v[i] = -(i*2+1);
      b_k[i] =  (i+1)*2;
      b_v[i] = -(i+1)*2;
      c_k[i] = c_k[i+N] = c_v[i] = c_v[i+N] = 0;
    }

    auto bufsz = tf::cuda_merge_buffer_size<decltype(p)>(N, N);
    tf::cudaDeviceVector<std::byte> buf(bufsz);

    tf::cuda_merge_by_key(
      p, 
      a_k, a_k+N, a_v, 
      b_k, b_k+N, b_v,
      c_k, c_v,
      tf::cuda_less<T>{}, 
      buf.data()
    );
    s.synchronize();

    for(int i=0; i<2*N; i++) {
      REQUIRE(c_k[i] == (i+1));
      REQUIRE(c_v[i] == -(i+1));
    }
    /*
    // ----------------- cudaFlow capturer
    // initialize the data
    for(int i=0; i<N; i++) {
      a_k[i] =  (i*2+1);
      a_v[i] = -(i*2+1);
      b_k[i] =  (i+1)*2;
      b_v[i] = -(i+1)*2;
      c_k[i] = c_k[i+N] = c_v[i] = c_v[i+N] = 0;
    }
    
    taskflow.emplace([&](){
      F cudaflow;
      cudaflow.merge_by_key(
        a_k, a_k+N, a_v, b_k, b_k+N, b_v, c_k, c_v, tf::cuda_less<T>{}
      );
      run_and_wait(cudaflow);
    });

    executor.run(taskflow).wait();
    
    for(int i=0; i<2*N; i++) {
      REQUIRE(c_k[i] == (i+1));
      REQUIRE(c_v[i] == -(i+1));
    }*/
    
    REQUIRE(hipFree(a_k) == hipSuccess);
    REQUIRE(hipFree(b_k) == hipSuccess);
    REQUIRE(hipFree(c_k) == hipSuccess);
    REQUIRE(hipFree(a_v) == hipSuccess);
    REQUIRE(hipFree(b_v) == hipSuccess);
    REQUIRE(hipFree(c_v) == hipSuccess);
  }
}

TEST_CASE("cudaflow.merge_keys_values.int" * doctest::timeout(300)) {
  merge_keys_values<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.merge_keys_values.float" * doctest::timeout(300)) {
  merge_keys_values<float, tf::cudaFlow>();
}

TEST_CASE("capturer.merge_keys_values.int" * doctest::timeout(300)) {
  merge_keys_values<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capturer.merge_keys_values.float" * doctest::timeout(300)) {
  merge_keys_values<float, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// sort
// ----------------------------------------------------------------------------

template <typename T, typename F>
void sort_keys() {
    
  tf::Executor executor;
  tf::Taskflow taskflow;

  for(int N=0; N<=1234567; N = N*2 + 1) {

    taskflow.clear();

    auto a = tf::cuda_malloc_shared<T>(N);
    tf::cudaStream s;
    auto p = tf::cudaDefaultExecutionPolicy{s};

    // ----------------- standalone asynchronous algorithms

    // initialize the data
    for(int i=0; i<N; i++) {
      a[i] = T(rand()%1000);
    }

    auto bufsz = tf::cuda_sort_buffer_size<decltype(p), T>(N);
    tf::cudaDeviceVector<std::byte> buf(bufsz);
    tf::cuda_sort(p, a, a+N, tf::cuda_less<T>{}, buf.data());
    s.synchronize();
    REQUIRE(std::is_sorted(a, a+N));

    /*
    // ----------------- cudaflow capturer
    for(int i=0; i<N; i++) {
      a[i] = T(rand()%1000);
    }
    
    taskflow.emplace([&](){
      F cudaflow;
      cudaflow.sort(a, a+N, tf::cuda_less<T>{});
      run_and_wait(cudaflow);
    });

    executor.run(taskflow).wait();

    REQUIRE(std::is_sorted(a, a+N)); */
    
    REQUIRE(hipFree(a) == hipSuccess);
  }
}

TEST_CASE("cudaflow.sort_keys.int" * doctest::timeout(300)) {
  sort_keys<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.sort_keys.float" * doctest::timeout(300)) {
  sort_keys<float, tf::cudaFlow>();
}

TEST_CASE("capture.sort_keys.int" * doctest::timeout(300)) {
  sort_keys<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.sort_keys.float" * doctest::timeout(300)) {
  sort_keys<float, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// sort key-value
// ----------------------------------------------------------------------------

template <typename T, typename F>
void sort_keys_values() {

  std::random_device rd;
  std::mt19937 g(rd());
    
  tf::Executor executor;
  tf::Taskflow taskflow;

  for(int N=1; N<=1234567; N = N*2 + 1) {

    taskflow.clear();

    auto a = tf::cuda_malloc_shared<T>(N);
    auto b = tf::cuda_malloc_shared<int>(N);

    tf::cudaStream s;
    auto p = tf::cudaDefaultExecutionPolicy{s};

    std::vector<int> indices(N);

    // ----------------- standalone asynchronous algorithms

    // initialize the data
    for(int i=0; i<N; i++) {
      a[i] = i;
      b[i] = i;
      indices[i] = i;
      //printf("a[%d]=%d, b[%d]=%d\n", i, a[i], i, b[i]);
    }
    std::shuffle(a, a+N, g);

    std::sort(indices.begin(), indices.end(), [&](auto i, auto j){
      return a[i] < a[j];
    });

    auto bufsz = tf::cuda_sort_buffer_size<decltype(p), T, int>(N);
    tf::cudaDeviceVector<std::byte> buf(bufsz);
    tf::cuda_sort_by_key(p, a, a+N, b, tf::cuda_less<T>{}, buf.data());
    s.synchronize();

    REQUIRE(std::is_sorted(a, a+N));
    for(int i=0; i<N; i++) {
      REQUIRE(indices[i] == b[i]);
    }

    /*// ----------------- cudaflow capturer
    // initialize the data
    for(int i=0; i<N; i++) {
      b[i] = i;
      indices[i] = i;
      //printf("a[%d]=%d, b[%d]=%d\n", i, a[i], i, b[i]);
    }
    std::shuffle(a, a+N, g);

    std::sort(indices.begin(), indices.end(), [&](auto i, auto j){
      return a[i] > a[j];
    });

    taskflow.emplace([&](){
      F cudaflow;
      cudaflow.sort_by_key(a, a+N, b, tf::cuda_greater<T>{});
      run_and_wait(cudaflow);
    });

    executor.run(taskflow).wait();

    REQUIRE(std::is_sorted(a, a+N, std::greater<T>{}));
    for(int i=0; i<N; i++) {
      REQUIRE(indices[i] == b[i]);
    }*/
    
    REQUIRE(hipFree(a) == hipSuccess);
    REQUIRE(hipFree(b) == hipSuccess);
  }
}

TEST_CASE("cudaflow.sort_keys_values.int" * doctest::timeout(300)) {
  sort_keys_values<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.sort_keys_values.float" * doctest::timeout(300)) {
  sort_keys_values<float, tf::cudaFlow>();
}

TEST_CASE("capture.sort_keys_values.int" * doctest::timeout(300)) {
  sort_keys_values<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.sort_keys_values.float" * doctest::timeout(300)) {
  sort_keys_values<float, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// find-if
// ----------------------------------------------------------------------------

template <typename T, typename F>
void find_if() {
    
  tf::Executor executor;
  tf::Taskflow taskflow;

  for(int N=0; N<=1234567; N += std::max(N/100, 1)) {

    taskflow.clear();

    auto a = tf::cuda_malloc_shared<T>(N);
    auto r = tf::cuda_malloc_shared<unsigned>(1);

    tf::cudaStream s;
    auto p = tf::cudaDefaultExecutionPolicy{s};
    
    // initialize the data
    for(int i=0; i<N; i++) {
      a[i] = i;
    }
    *r = 1234;

    // ----------------- standalone asynchronous algorithms

    tf::cuda_find_if(p, a, a+N, r, []__device__(int v){ return v == 5000; });
    s.synchronize();

    if(N <= 5000) {
      REQUIRE(*r == N);
    }
    else {
      REQUIRE(*r == 5000);
    }

    // ----------------- cudaflow capturer
    *r = 1234;
    
    taskflow.emplace([&](){
      F cudaflow;
      cudaflow.find_if(a, a+N, r, []__device__(int v){ return v == 5000; });
      run_and_wait(cudaflow);
    });

    executor.run(taskflow).wait();
    
    if(N <= 5000) {
      REQUIRE(*r == N);
    }
    else {
      REQUIRE(*r == 5000);
    }
    
    REQUIRE(hipFree(a) == hipSuccess);
    REQUIRE(hipFree(r) == hipSuccess);
  }
}

TEST_CASE("cudaflow.find_if.int" * doctest::timeout(300)) {
  find_if<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.find_if.float" * doctest::timeout(300)) {
  find_if<float, tf::cudaFlow>();
}

TEST_CASE("capture.find_if.int" * doctest::timeout(300)) {
  find_if<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capture.find_if.float" * doctest::timeout(300)) {
  find_if<float, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// min_element
// ----------------------------------------------------------------------------

template <typename T, typename F>
void min_element() {
    
  tf::Executor executor;
  tf::Taskflow taskflow;

  for(int N=0; N<=1234567; N += std::max(N/10, 1)) {

    taskflow.clear();

    auto a = tf::cuda_malloc_shared<T>(N);
    auto r = tf::cuda_malloc_shared<unsigned>(1);
    auto min = std::numeric_limits<T>::max();
    
    tf::cudaStream s;
    auto p = tf::cudaDefaultExecutionPolicy{s};

    // initialize the data
    for(int i=0; i<N; i++) {
      a[i] = rand();
      min = std::min(min, a[i]);
    }
    *r = 1234;

    // ----------------- standalone asynchronous algorithms

    tf::cudaDeviceVector<std::byte> buf(
      tf::cuda_min_element_buffer_size<decltype(p), T>(N)
    );

    tf::cuda_min_element(
      p, a, a+N, r, tf::cuda_less<T>{}, buf.data()
    );
    s.synchronize();

    if(min != std::numeric_limits<T>::max()) {
      REQUIRE(a[*r] == min);
    }
    else {
      REQUIRE(*r == N);
    }
    
    // ----------------- cudaflow
    *r = 1234;
    
    taskflow.emplace([&](){
      F cudaflow;
      cudaflow.min_element(a, a+N, r, tf::cuda_less<T>{});
      run_and_wait(cudaflow);
    });

    executor.run(taskflow).wait();
    
    if(min != std::numeric_limits<T>::max()) {
      REQUIRE(a[*r] == min);
    }
    else {
      REQUIRE(*r == N);
    }
    
    REQUIRE(hipFree(a) == hipSuccess);
    REQUIRE(hipFree(r) == hipSuccess);
  }
}

TEST_CASE("cudaflow.min_element.int" * doctest::timeout(300)) {
  min_element<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.min_element.float" * doctest::timeout(300)) {
  min_element<float, tf::cudaFlow>();
}

TEST_CASE("capturer.min_element.int" * doctest::timeout(300)) {
  min_element<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capturer.min_element.float" * doctest::timeout(300)) {
  min_element<float, tf::cudaFlowCapturer>();
}

// ----------------------------------------------------------------------------
// max_element
// ----------------------------------------------------------------------------

template <typename T, typename F>
void max_element() {
    
  tf::Executor executor;
  tf::Taskflow taskflow;

  for(int N=0; N<=1234567; N += std::max(N/10, 1)) {

    taskflow.clear();

    auto a = tf::cuda_malloc_shared<T>(N);
    auto r = tf::cuda_malloc_shared<unsigned>(1);
    auto max = std::numeric_limits<T>::lowest();
    
    tf::cudaStream s;
    auto p = tf::cudaDefaultExecutionPolicy{s};

    // initialize the data
    for(int i=0; i<N; i++) {
      a[i] = rand();
      max = std::max(max, a[i]);
    }
    *r = 1234;

    // ----------------- standalone asynchronous algorithms

    tf::cudaDeviceVector<std::byte> buf(
      tf::cuda_max_element_buffer_size<decltype(p), T>(N)
    );

    tf::cuda_max_element(p, a, a+N, r, tf::cuda_less<T>{}, buf.data());
    s.synchronize();

    if(max != std::numeric_limits<T>::lowest()) {
      REQUIRE(a[*r] == max);
    }
    else {
      REQUIRE(*r == N);
    }
    
    // ----------------- cudaflow
    *r = 1234;
    
    taskflow.emplace([&](){
      F cudaflow;
      cudaflow.max_element(a, a+N, r, tf::cuda_less<T>{});
      run_and_wait(cudaflow);
    });

    executor.run(taskflow).wait();
    
    if(max != std::numeric_limits<T>::lowest()) {
      REQUIRE(a[*r] == max);
    }
    else {
      REQUIRE(*r == N);
    }
    
    REQUIRE(hipFree(a) == hipSuccess);
    REQUIRE(hipFree(r) == hipSuccess);
  }
}

TEST_CASE("cudaflow.max_element.int" * doctest::timeout(300)) {
  max_element<int, tf::cudaFlow>();
}

TEST_CASE("cudaflow.max_element.float" * doctest::timeout(300)) {
  max_element<float, tf::cudaFlow>();
}

TEST_CASE("capturer.max_element.int" * doctest::timeout(300)) {
  max_element<int, tf::cudaFlowCapturer>();
}

TEST_CASE("capturer.max_element.float" * doctest::timeout(300)) {
  max_element<float, tf::cudaFlowCapturer>();
}

/*// --------------------------------------------------------------------------
// row-major transpose
// ----------------------------------------------------------------------------

// Disable for now - better to use cublasFlowCapturer

template <typename T>
__global__
void verify(const T* din_mat, const T* dout_mat, bool* check, size_t rows, size_t cols) {
  
  size_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  size_t size = rows * cols;
  for(; tid < size; tid += gridDim.x * blockDim.x) {
    if(din_mat[tid] != dout_mat[tid / cols + (tid % cols) * rows]) {
      *check = false;
      return;
    }
  }
}

template <typename T>
void transpose() {
  tf::Executor executor;

  for(size_t rows = 1; rows <= 7999; rows*=2+3) {
    for(size_t cols = 1; cols <= 8021; cols*=3+5) {

      tf::Taskflow taskflow;
      std::vector<T> hinput_mat(rows * cols);

      std::generate_n(hinput_mat.begin(), rows * cols, [](){ return ::rand(); });

      T* dinput_mat {nullptr};
      T* doutput_mat {nullptr};
      bool* check {nullptr};
      
       //allocate
      auto allocate = taskflow.emplace([&]() {
        REQUIRE(hipMalloc(&dinput_mat, (rows * cols) * sizeof(T)) == hipSuccess);
        REQUIRE(hipMalloc(&doutput_mat, (rows * cols) * sizeof(T)) == hipSuccess);
        REQUIRE(hipMallocManaged(&check, sizeof(bool)) == hipSuccess);
        *check = true;
      }).name("allocate");

       //transpose
      auto cudaflow = taskflow.emplace([&](tf::cudaFlow& cf) {
        auto h2d_input_t = cf.copy(dinput_mat, hinput_mat.data(), rows * cols).name("h2d");

        auto kernel_t = tf::cudaBLAF(cf).transpose(
          dinput_mat,
          doutput_mat,
          rows,
          cols
        );

        auto verify_t = cf.kernel(
          32,
          512,
          0,
          verify<T>,
          dinput_mat,
          doutput_mat,
          check,
          rows,
          cols
        );

        h2d_input_t.precede(kernel_t);
        kernel_t.precede(verify_t);
      }).name("transpose");


       //free memory
      auto deallocate = taskflow.emplace([&](){
        REQUIRE(hipFree(dinput_mat) == hipSuccess);
        REQUIRE(hipFree(doutput_mat) == hipSuccess);
      }).name("deallocate");
      

      allocate.precede(cudaflow);
      cudaflow.precede(deallocate);

      executor.run(taskflow).wait();
      REQUIRE(*check);
    }
  }
}

TEST_CASE("transpose.int" * doctest::timeout(300) ) {
  transpose<int>();
}

TEST_CASE("transpose.float" * doctest::timeout(300) ) {
  transpose<float>();
}


TEST_CASE("transpose.double" * doctest::timeout(300) ) {
  transpose<double>();
}

// ----------------------------------------------------------------------------
// row-major matrix multiplication
// ----------------------------------------------------------------------------

template <typename T>
void matmul() {
  tf::Taskflow taskflow;
  tf::Executor executor;
  
  std::vector<T> a, b, c;

  for(int m=1; m<=1992; m=2*m+1) {
    for(int k=1; k<=1012; k=2*k+3) {
      for(int n=1; n<=1998; n=2*n+8) {

        taskflow.clear();

        T* ha {nullptr};
        T* hb {nullptr};
        T* hc {nullptr};
        T* da {nullptr};
        T* db {nullptr};
        T* dc {nullptr};
      
        T val_a = ::rand()%5-1;
        T val_b = ::rand()%7-3;

        auto hosta = taskflow.emplace([&](){ 
          a.resize(m*k);
          std::fill_n(a.begin(), m*k, val_a);
          ha = a.data();
          REQUIRE(hipMalloc(&da, m*k*sizeof(T)) == hipSuccess);
        }).name("ha");

        auto hostb = taskflow.emplace([&](){ 
          b.resize(k*n);
          std::fill_n(b.begin(), k*n, val_b);
          hb = b.data();
          REQUIRE(hipMalloc(&db, k*n*sizeof(T)) == hipSuccess);
        }).name("hb");

        auto hostc = taskflow.emplace([&](){
          c.resize(m*n);
          hc = c.data();
          REQUIRE(hipMalloc(&dc, m*n*sizeof(T)) == hipSuccess);
        }).name("hc");

        auto cuda = taskflow.emplace([&](tf::cudaFlow& cf){
          auto pa = cf.copy(da, ha, m*k);
          auto pb = cf.copy(db, hb, k*n);

          auto op = tf::cudaBLAF(cf).matmul(
            da, db, dc, m, k, n 
          ).name("op");

          auto cc = cf.copy(hc, dc, m*n).name("cc");

          op.precede(cc).succeed(pa, pb);
        });

        cuda.succeed(hosta, hostb, hostc);

        executor.run(taskflow).wait();

        int ans = val_a*val_b*k;
        for(const auto& x : c) {
          REQUIRE((int)x == ans);
        }

        REQUIRE(hipFree(da) == hipSuccess);
        REQUIRE(hipFree(db) == hipSuccess);
        REQUIRE(hipFree(dc) == hipSuccess);
      }
    }
  }
}

TEST_CASE("matmul.int" * doctest::timeout(300) ) {
  matmul<int>();
}

TEST_CASE("matmul.float" * doctest::timeout(300) ) {
  matmul<float>();
}

TEST_CASE("matmul.double" * doctest::timeout(300) ) {
  matmul<double>();
}*/


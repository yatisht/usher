#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>
#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cudaflow.hpp>
#include <taskflow/cuda/algorithm/scan.hpp>

// ----------------------------------------------------------------------------
// cuda_scan
// ----------------------------------------------------------------------------

template <typename T>
void cuda_scan() {

  tf::Taskflow taskflow;
  tf::Executor executor;
  
  for(int n=0; n<=1234567; n = (n<=100) ? n+1 : n*2 + 1) {

    taskflow.emplace([n](){
  
      auto data1 = tf::cuda_malloc_shared<int>(n);
      auto data2 = tf::cuda_malloc_shared<int>(n);
      auto scan1 = tf::cuda_malloc_shared<int>(n);
      auto scan2 = tf::cuda_malloc_shared<int>(n);

      // --------------------------------------------------------------------------
      // inclusive/exclusive scan
      // --------------------------------------------------------------------------

      // initialize the data
      std::iota(data1, data1 + n, 0);
      std::iota(data2, data2 + n, 0);
      
      tf::cudaStream stream;
      tf::cudaDefaultExecutionPolicy policy(stream);

      // declare the buffer
      void* buff;
      hipMalloc(&buff, policy.scan_bufsz<int>(n));
      
      // create inclusive and exclusive scan tasks
      tf::cuda_inclusive_scan(policy, data1, data1+n, scan1, tf::cuda_plus<int>{}, buff);
      tf::cuda_exclusive_scan(policy, data2, data2+n, scan2, tf::cuda_plus<int>{}, buff);

      stream.synchronize();
      
      // inspect 
      for(int i=1; i<n; i++) {
        REQUIRE(scan1[i] == (scan1[i-1] + data1[i]));
        REQUIRE(scan2[i] == (scan2[i-1] + data2[i-1]));
      }
  
      // deallocate the data
      REQUIRE(hipFree(data1) == hipSuccess);
      REQUIRE(hipFree(data2) == hipSuccess);
      REQUIRE(hipFree(scan1) == hipSuccess);
      REQUIRE(hipFree(scan2) == hipSuccess);
      REQUIRE(hipFree(buff)  == hipSuccess);
    });
  }

  executor.run(taskflow).wait();
}

TEST_CASE("cuda_scan.int" * doctest::timeout(300)) {
  cuda_scan<int>();
}

// ----------------------------------------------------------------------------
// transform_scan
// ----------------------------------------------------------------------------

template <typename T>
void cuda_transform_scan() {

  tf::Taskflow taskflow;
  tf::Executor executor;
  
  for(int n=0; n<=1234567; n = (n<=100) ? n+1 : n*2 + 1) {
  
    taskflow.emplace([n](){

      auto data1 = tf::cuda_malloc_shared<int>(n);
      auto data2 = tf::cuda_malloc_shared<int>(n);
      auto scan1 = tf::cuda_malloc_shared<int>(n);
      auto scan2 = tf::cuda_malloc_shared<int>(n);

      // --------------------------------------------------------------------------
      // inclusive/exclusive scan
      // --------------------------------------------------------------------------

      tf::cudaStream stream;
      tf::cudaDefaultExecutionPolicy policy(stream);

      // declare the buffer
      void* buff;
      hipMalloc(&buff, policy.scan_bufsz<int>(n));
      
      // initialize the data
      std::iota(data1, data1 + n, 0);
      std::iota(data2, data2 + n, 0);
      
      // transform inclusive scan
      tf::cuda_transform_inclusive_scan(policy,
        data1, data1+n, scan1, tf::cuda_plus<int>{},
        [] __device__ (int a) { return a*10; },
        buff
      );

      // transform exclusive scan
      tf::cuda_transform_exclusive_scan(policy,
        data2, data2+n, scan2, tf::cuda_plus<int>{},
        [] __device__ (int a) { return a*11; },
        buff
      );
      
      stream.synchronize();
  
      // inspect 
      for(int i=1; i<n; i++) {
        REQUIRE(scan1[i] == scan1[i-1] + data1[i] * 10);
        REQUIRE(scan2[i] == scan2[i-1] + data2[i-1] * 11);
      }
  
      // deallocate the data
      REQUIRE(hipFree(data1) == hipSuccess);
      REQUIRE(hipFree(data2) == hipSuccess);
      REQUIRE(hipFree(scan1) == hipSuccess);
      REQUIRE(hipFree(scan2) == hipSuccess);
      REQUIRE(hipFree(buff)  == hipSuccess);
    });
  }

  executor.run(taskflow).wait();
}

TEST_CASE("cuda_transform_scan.int" * doctest::timeout(300)) {
  cuda_transform_scan<int>();
}

#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN

#include <doctest.h>
#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cudaflow.hpp>
#include <taskflow/cuda/algorithm/for_each.hpp>

#include "./details/graph_executor.hpp"
#include "./details/tree.hpp"
#include "./details/random_DAG.hpp"
#include "./details/tree.hpp"
#include "./details/diamond.hpp"

// ----------------------------------------------------------------------------
// Graph traversal
// ----------------------------------------------------------------------------
template <typename GRAPH, typename OPT, typename... OPT_Args>
void traversal(OPT_Args&&... args) {
  for(int i = 0; i < 13; ++i) {
    Graph* g;
    if constexpr(std::is_same_v<GRAPH, Tree>) {
      g = new Tree(::rand() % 3 + 1, ::rand() % 5 + 1);
    }
    else if constexpr(std::is_same_v<GRAPH, RandomDAG>) {
      g = new RandomDAG(::rand() % 10 + 1, ::rand() % 10 + 1, ::rand() % 10 + 1);
    }
    else if constexpr(std::is_same_v<GRAPH, Diamond>) {
      g = new Diamond(::rand() % 10 + 1, ::rand() % 10 + 1);
    }
    GraphExecutor<OPT> executor(*g, 0); 
    executor.traversal(std::forward<OPT_Args>(args)...);

    REQUIRE(g->traversed());
    delete g;
  }

}

TEST_CASE("cudaFlowCapturer.tree.Sequential") {
  traversal<Tree, tf::cudaFlowSequentialOptimizer>();
}

TEST_CASE("cudaFlowCapturer.tree.RoundRobin.1") {
  traversal<Tree, tf::cudaFlowRoundRobinOptimizer>(1);
}

TEST_CASE("cudaFlowCapturer.tree.RoundRobin.2") {
  traversal<Tree, tf::cudaFlowRoundRobinOptimizer>(2);
}

TEST_CASE("cudaFlowCapturer.tree.RoundRobin.3") {
  traversal<Tree, tf::cudaFlowRoundRobinOptimizer>(3);
}

TEST_CASE("cudaFlowCapturer.tree.RoundRobin.4") {
  traversal<Tree, tf::cudaFlowRoundRobinOptimizer>(4);
}

//TEST_CASE("cudaFlowCapturer.tree.Greedy.1") {
//  traversal<Tree, tf::cudaGreedyCapturing>(1);
//}
//
//TEST_CASE("cudaFlowCapturer.tree.Greedy.2") {
//  traversal<Tree, tf::cudaGreedyCapturing>(2);
//}
//
//TEST_CASE("cudaFlowCapturer.tree.Greedy.3") {
//  traversal<Tree, tf::cudaGreedyCapturing>(3);
//}
//
//TEST_CASE("cudaFlowCapturer.tree.Greedy.4") {
//  traversal<RandomDAG, tf::cudaGreedyCapturing>(4);
//}

TEST_CASE("cudaFlowCapturer.randomDAG.Sequential") {
  traversal<RandomDAG,tf::cudaFlowSequentialOptimizer>();
}

TEST_CASE("cudaFlowCapturer.randomDAG.RoundRobin.1") {
  traversal<RandomDAG, tf::cudaFlowRoundRobinOptimizer>(1);
}

TEST_CASE("cudaFlowCapturer.randomDAG.RoundRobin.2") {
  traversal<RandomDAG, tf::cudaFlowRoundRobinOptimizer>(2);
}

TEST_CASE("cudaFlowCapturer.randomDAG.RoundRobin.3") {
  traversal<RandomDAG, tf::cudaFlowRoundRobinOptimizer>(3);
}

TEST_CASE("cudaFlowCapturer.randomDAG.RoundRobin.4") {
  traversal<RandomDAG, tf::cudaFlowRoundRobinOptimizer>(4);
}

//TEST_CASE("cudaFlowCapturer.randomDAG.Greedy.1") {
//  traversal<RandomDAG, tf::cudaGreedyCapturing>(1);
//}
//
//TEST_CASE("cudaFlowCapturer.randomDAG.Greedy.2") {
//  traversal<RandomDAG, tf::cudaGreedyCapturing>(2);
//}
//
//TEST_CASE("cudaFlowCapturer.randomDAG.Greedy.3") {
//  traversal<RandomDAG, tf::cudaGreedyCapturing>(3);
//}
//
//TEST_CASE("cudaFlowCapturer.randomDAG.Greedy.4") {
//  traversal<RandomDAG, tf::cudaGreedyCapturing>(4);
//}

TEST_CASE("cudaFlowCapturer.diamond.Sequential") {
  traversal<Diamond, tf::cudaFlowSequentialOptimizer>();
}

TEST_CASE("cudaFlowCapturer.diamond.RoundRobin.1") {
  traversal<Diamond, tf::cudaFlowRoundRobinOptimizer>(1);
}

TEST_CASE("cudaFlowCapturer.diamond.RoundRobin.2") {
  traversal<Diamond, tf::cudaFlowRoundRobinOptimizer>(2);
}

TEST_CASE("cudaFlowCapturer.diamond.RoundRobin.3") {
  traversal<Diamond, tf::cudaFlowRoundRobinOptimizer>(3);
}

TEST_CASE("cudaFlowCapturer.diamond.RoundRobin.4") {
  traversal<Diamond, tf::cudaFlowRoundRobinOptimizer>(4);
}

//TEST_CASE("cudaFlowCapturer.diamond.Greedy.1") {
//  traversal<Diamond, tf::cudaGreedyCapturing>(1);
//}
//
//TEST_CASE("cudaFlowCapturer.diamond.Greedy.2") {
//  traversal<Diamond, tf::cudaGreedyCapturing>(2);
//}
//
//TEST_CASE("cudaFlowCapturer.diamond.Greedy.3") {
//  traversal<Diamond, tf::cudaGreedyCapturing>(3);
//}
//
//TEST_CASE("cudaFlowCapturer.diamond.Greedy.4") {
//  traversal<Diamond, tf::cudaGreedyCapturing>(4);
//}

//------------------------------------------------------
// dependencies
//------------------------------------------------------

template <typename OPT, typename... OPT_Args>
void dependencies(OPT_Args ...args) {
  
  for(int t = 0; t < 17; ++t) {
    int num_partitions = ::rand() % 5 + 1;
    int num_iterations = ::rand() % 7 + 1;

    Diamond g(num_partitions, num_iterations);

    tf::cudaFlowCapturer cf;
    cf.make_optimizer<OPT>(std::forward<OPT_Args>(args)...);

    int* inputs{nullptr};
    REQUIRE(hipMallocManaged(&inputs, num_partitions * sizeof(int)) == hipSuccess);
    REQUIRE(hipMemset(inputs, 0, num_partitions * sizeof(int)) == hipSuccess);

    std::vector<std::vector<tf::cudaTask>> tasks;
    tasks.resize(g.get_size());

    for(size_t l = 0; l < g.get_size(); ++l) {
      tasks[l].resize((g.get_graph())[l].size());
      for(size_t i = 0; i < (g.get_graph())[l].size(); ++i) {
        
        if(l % 2 == 1) {
          tasks[l][i] = cf.single_task([inputs, i] __device__ () {
            inputs[i]++;
          });
        }
        else {
          tasks[l][i] = cf.on([=](hipStream_t stream){
            cuda_for_each(
              tf::cudaDefaultExecutionPolicy(stream), inputs, inputs + num_partitions, 
              [] __device__ (int& v) { v*=2; }
            );
          });
        }
      }
    }

    for(size_t l = 0; l < g.get_size() - 1; ++l) {
      for(size_t i = 0; i < (g.get_graph())[l].size(); ++i) {
        for(auto&& out_node: g.at(l, i).out_nodes) {
          tasks[l][i].precede(tasks[l + 1][out_node]);
        }
      }
    }

    tf::cudaStream stream;
    cf.run(stream);
    stream.synchronize();
    
    int result = 2;
    for(int i = 1; i < num_iterations; ++i) {
      result = result * 2 + 2;
    }

    for(int i = 0; i < num_partitions; ++i) {
      REQUIRE(inputs[i] == result);
    }

    REQUIRE(hipFree(inputs) == hipSuccess);
  }
}

TEST_CASE("cudaFlowCapturer.dependencies.diamond.Sequential") {
  dependencies<tf::cudaFlowSequentialOptimizer>();
}

TEST_CASE("cudaFlowCapturer.dependencies.diamond.RoundRobin.1") {
  dependencies<tf::cudaFlowRoundRobinOptimizer>(1);
}

TEST_CASE("cudaFlowCapturer.dependencies.diamond.RoundRobin.2") {
  dependencies<tf::cudaFlowRoundRobinOptimizer>(2);
}

TEST_CASE("cudaFlowCapturer.dependencies.diamond.RoundRobin.3") {
  dependencies<tf::cudaFlowRoundRobinOptimizer>(3);
}

TEST_CASE("cudaFlowCapturer.dependencies.diamond.RoundRobin.4") {
  dependencies<tf::cudaFlowRoundRobinOptimizer>(4);
}

//TEST_CASE("cudaFlowCapturer.dependencies.diamond.Greedy.1") {
//  dependencies<tf::cudaGreedyCapturing>(1);
//}
//
//TEST_CASE("cudaFlowCapturer.dependencies.diamond.Greedy.2") {
//  dependencies<tf::cudaGreedyCapturing>(2);
//}
//
//TEST_CASE("cudaFlowCapturer.dependencies.diamond.Greedy.3") {
//  dependencies<tf::cudaGreedyCapturing>(3);
//}
//
//TEST_CASE("cudaFlowCapturer.dependencies.diamond.Greedy.4") {
//  dependencies<tf::cudaGreedyCapturing>(4);
//}

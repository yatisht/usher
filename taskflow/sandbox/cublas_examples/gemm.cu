// This program performs general matrix multiplication on row-major layout
// using tf::cublasFlowCapturer::c_gemm.

#include <taskflow/taskflow.hpp>
#include <taskflow/cudaflow.hpp>
#include <taskflow/cublasflow.hpp>

int main() {

  const int M = 2, N = 4, K = 3;

  const std::vector<float> hA = {
    11, 12, 13, 
    14, 15, 16
  };  // M x K

  const std::vector<float> hB = {
    11, 12, 13, 14,
    15, 16, 17, 18,
    19, 20, 21, 22
  };  // K x N

  const std::vector<float> golden = {
    548, 584, 620, 656,
    683, 728, 773, 818 
  };  //  M x N

  std::vector<float> hC(M*N);
    
  //auto dA = tf::cuda_malloc_device<float>(hA.size());
  //auto dB = tf::cuda_malloc_device<float>(hB.size());
  //auto dC = tf::cuda_malloc_device<float>(hC.size());
  //auto dAlpha = tf::cuda_malloc_device<float>(1);
  //auto dBeta  = tf::cuda_malloc_device<float>(1);
  float *dA, *dB, *dC, *dAlpha, *dBeta;

  tf::Taskflow taskflow("Matrix Multiplication");
  tf::Executor executor;

  auto malloc_dA = taskflow.emplace(
    [&](){ dA = tf::cuda_malloc_device<float>(hA.size()); }
  ).name("malloc_dA");
  
  auto malloc_dB = taskflow.emplace(
    [&](){ dB = tf::cuda_malloc_device<float>(hB.size()); }
  ).name("malloc_dB");
  
  auto malloc_dC = taskflow.emplace(
    [&](){ dC = tf::cuda_malloc_device<float>(hC.size()); }
  ).name("malloc_dC");
  
  auto malloc_dAlpha = taskflow.emplace(
    [&](){ dAlpha = tf::cuda_malloc_device<float>(1); }
  ).name("malloc_dAlpha");
  
  auto malloc_dBeta = taskflow.emplace(
    [&](){ dBeta = tf::cuda_malloc_device<float>(1); }
  ).name("malloc_dBeta");

  auto cublasFlow = taskflow.emplace([&](tf::cudaFlowCapturer& capturer) {
    auto blas  = capturer.make_capturer<tf::cublasFlowCapturer>();

    auto alpha = capturer.single_task([=] __device__ () { *dAlpha = 1; })
                         .name("alpha=1");
    auto beta  = capturer.single_task([=] __device__ () { *dBeta  = 0; })
                         .name("beta=0");
    auto copyA = capturer.copy(dA, hA.data(), hA.size()).name("copyA"); 
    auto copyB = capturer.copy(dB, hB.data(), hB.size()).name("copyB");
    auto gemm  = blas->c_gemm(HIPBLAS_OP_N, HIPBLAS_OP_N,
      M, N, K, dAlpha, dA, K, dB, N, dBeta, dC, N
    ).name("C = alpha * A * B + beta * C");
    auto copyC = capturer.copy(hC.data(), dC, hC.size()).name("copyC");

    gemm.succeed(alpha, beta, copyA, copyB)
        .precede(copyC);

    capturer.dump(std::cout);  // dump the graph constructed so far.
  }).name("cublasFlow");

  cublasFlow.succeed(
    malloc_dA, malloc_dB, malloc_dC, malloc_dAlpha, malloc_dBeta
  );

  executor.run(taskflow).wait();

  taskflow.dump(std::cout);
  
  std::cout << "Matrix C:\n";
  for(int m=0; m<M; m++) {
    for(int n=0; n<N; n++) {
      std::cout << hC[m*N+n] << ' ';
    }
    std::cout << '\n';
  }

  return 0;
}







